#include "hip/hip_runtime.h"
/*

   lv2_gpu_sort.cu       sort lev2 with GPU

#    Copyright (C) 2015, The University of Hong Kong.
#
#    This program is free software; you can redistribute it and/or
#    modify it under the terms of the GNU General Public License
#    as published by the Free Software Foundation; either version 3
#    of the License, or (at your option) any later version.
#
#    This program is distributed in the hope that it will be useful,
#    but WITHOUT ANY WARRANTY; without even the implied warranty of
#    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
#    GNU General Public License for more details.
#
#    You should have received a copy of the GNU General Public License
#    along with this program; if not, write to the Free Software
#    Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  03110-1301, USA.

    Date   : 1st Jan 2015
    Author : Binghang LIU
    Change : Generate this file by fishing from Chi Man LIU's code.
*/

#include <stdio.h>
#include <assert.h>
#include <stdarg.h>
#include <stdlib.h>

#include <omp.h>
#include <pthread.h>
#include <assert.h>
#include <stdarg.h>
#include <vector>
#include <algorithm>


#include "lv2_gpu_sort.h"

#include "b40c/radix_sort/enactor.cuh"
#include "b40c/util/multiple_buffering.cuh"

size_t get_gpu_mem()
{
    size_t free_gpu_mem, total_gpu_mem;
    assert(hipMemGetInfo(&free_gpu_mem, &total_gpu_mem) == hipSuccess);
    fprintf(stderr, "Free GPU mem: %lld\n", free_gpu_mem);
    return free_gpu_mem;
}

__global__ void permutation_kernel( uint32_t* index, uint32_t* val, uint32_t* new_val, int num_elements ) {
    int tid = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

    if (tid < num_elements)
      new_val[tid] = val[index[tid] & 0x1FFFFFFF];
}


void lv2_gpu_sort(uint32_t *lv2_substrings, uint32_t *permutation, int uint32_ts_per_substring, int64_t width, int64_t lv2_num_items){

	uint32_t *d_keys, *d_values;
        assert(hipMalloc( (void**) &d_keys, sizeof(uint32_t) * lv2_num_items ) == hipSuccess);
        assert(hipMalloc( (void**) &d_values, sizeof(uint32_t) * lv2_num_items ) == hipSuccess);
        b40c::radix_sort::Enactor enactor;
        b40c::util::DoubleBuffer<uint32_t, uint32_t> ss(d_keys, d_values);

        assert(hipMemcpy( ss.d_values[ss.selector], permutation, sizeof(uint32_t) * lv2_num_items, hipMemcpyHostToDevice ) == hipSuccess);
        for (int iteration = uint32_ts_per_substring - 1; iteration >= 0; --iteration ) { // TODO uint32_ts_per_suffix ?
        	if ( iteration == uint32_ts_per_substring - 1 ) { // TODO uint32_ts_per_suffix
                	assert(hipMemcpy( ss.d_keys[ss.selector], lv2_substrings + ( iteration * width ),
                        sizeof(uint32_t) * lv2_num_items, hipMemcpyHostToDevice ) == hipSuccess);
                } else {
                	assert(hipMemcpy( ss.d_keys[1-ss.selector], lv2_substrings + ( iteration * width ),
                        sizeof(uint32_t) * lv2_num_items, hipMemcpyHostToDevice ) == hipSuccess);
                        int num_blocks = ( lv2_num_items + THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK;

                        permutation_kernel<<<num_blocks, THREADS_PER_BLOCK>>>( ss.d_values[ss.selector], ss.d_keys[1-ss.selector],
                                                                               ss.d_keys[ss.selector], lv2_num_items );
                }
                assert(enactor.Sort<b40c::radix_sort::LARGE_SIZE>( ss, lv2_num_items ) == hipSuccess);
        }

        // free device memory EXCEPT sort_indexes
        if (ss.d_keys[ss.selector]) hipFree(ss.d_keys[ss.selector]);
        if (ss.d_keys[1-ss.selector]) hipFree(ss.d_keys[1-ss.selector]);
        if (ss.d_values[1-ss.selector]) hipFree(ss.d_values[1-ss.selector]);
        ///////////////////// END GPU SORT ////////////////////////////
        assert(hipMemcpy( permutation, ss.d_values[ss.selector], sizeof(int) * lv2_num_items, hipMemcpyDeviceToHost ) == hipSuccess);
	if (ss.d_values[ss.selector]) hipFree(ss.d_values[ss.selector]);
}

